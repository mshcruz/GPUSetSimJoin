#include "hip/hip_runtime.h"
#include "stdio.h"
#include "kernel.h"
#include "hiprand/hiprand_kernel.h"
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

#define THREADS_PER_BLOCK 256

__device__ int d_similarPairsCount(0);

__device__ unsigned int
MurmurHash2 ( const void * key, int len, unsigned int seed )
{
  // 'm' and 'r' are mixing constants generated offline.
  // They're not really 'magic', they just happen to work well.

  const unsigned int m = 0x5bd1e995;
  const int r = 24;

  // Initialize the hash to a 'random' value

  unsigned int h = seed ^ len;

  // Mix 4 bytes at a time into the hash

  const unsigned char * data = (const unsigned char *)key;

  while(len >= 4)
    {
      unsigned int k = *(unsigned int *)data;

      k *= m;
      k ^= k >> r;
      k *= m;

      h *= m;
      h ^= k;

      data += 4;
      len -= 4;
    }

  // Handle the last few bytes of the input array

  switch(len)
    {
    case 3: h ^= data[2] << 16;
    case 2: h ^= data[1] << 8;
    case 1: h ^= data[0];
      h *= m;
    };

  // Do a few final mixes of the hash to ensure the last few
  // bytes are well-incorporated.

  h ^= h >> 13;
  h *= m;
  h ^= h >> 15;

  return h;
}

// __global__ void
// computeSignatureMatrix_kernel(int *d_signatureMatrix, int *d_cmRowIdx, int* d_cmColPtr, int numShingles, int numSets, int numBins)
// {
//   const int tid = threadIdx.x + blockDim.x * blockIdx.x;
//   int binSize = numShingles/numBins;
//   if (numShingles % numBins) binSize++;

//   if (tid < numSets) {
//     for (int i = 0; i < numBins; i++) {
//       d_signatureMatrix[i + (tid*numBins)] = INT_MAX; //NOT Coalesced
//     }
//     for (int i = d_cmColPtr[tid]; i < d_cmColPtr[tid+1]; i++) { //Coalesced
//       int shingleIdx = d_cmRowIdx[i]; //Not coalesced
//       int shingleNewIdx = MurmurHash2(&shingleIdx, 4, 0)%numShingles; //To do: remove mod
//       int binIdx = shingleNewIdx/binSize;
//       int offSetSM = binIdx + (tid*numBins);
//        d_signatureMatrix[offSetSM] = min(d_signatureMatrix[offSetSM], shingleNewIdx); //Not coalesced
//     }
//   }
// }

__global__ void
computeSignatureMatrix_kernel(int *d_signatureMatrix, int *d_cmRowIdx, int* d_cmColPtr, int numShingles, int rSize, int sSize)
{
  __shared__ int s_signatures[NUM_BINS];
  // __shared__ int s_initPosition;
  // __shared__ int s_endPosition;
  int binSize = numShingles/NUM_BINS;
  if (numShingles % NUM_BINS) binSize++;

  for (int t = blockIdx.x; t < (rSize+sSize); t += gridDim.x) { //Each block computes the signature for one set
    __syncthreads(); // Added by Kozawa
                     // Without this synchronization, s_signatures may be updated
                     // before the copy to device memory completes.
    //1 - Initialize the signatures in the shared memory
    for (int i = threadIdx.x; i < NUM_BINS; i += blockDim.x) {
      s_signatures[i] = INT_MAX;
    }
    __syncthreads();

    //2 - Get the positions of the elements for this set in the rowInd array
    int begin = d_cmColPtr[t];
    int end = d_cmColPtr[t + 1];

    /*** __syncthreads is necessary if you use shared memory. (by Kozawa) ***/
    // if (threadIdx.x == 0) {
    //   s_initPosition = d_cmColPtr[t];
    // }
    // if (threadIdx.x == 1) {
    //   s_endPosition = d_cmColPtr[t+1];
    // }
    // int numShinglesInRecord = s_endPosition - s_initPosition;

    int numShinglesInRecord = end - begin;

    //3 - Apply the hash function on the element and get its new position
    for (int i = threadIdx.x; i < numShinglesInRecord; i += blockDim.x) {
      int shingleIdx = d_cmRowIdx[begin + i]; //Coalesced
      int hashedShingleIdx = MurmurHash2(&shingleIdx, 4, 0)%numShingles;
      int binIdx = hashedShingleIdx / binSize;
      //Keep the minimum value for that position
      atomicMin(&s_signatures[binIdx], hashedShingleIdx); //Atomic
    }
    __syncthreads();

    //4 - Copy signature to signature matrix in the device memory
    for (int i = threadIdx.x; i < NUM_BINS; i += blockDim.x) {
      d_signatureMatrix[t * NUM_BINS + i] = s_signatures[i]; //Coalesced
    }
  }
}

__global__ void transpose  // M-by-N to N-by-M
(const int *src, int *dst, int M, int N)
{
  __shared__ int s_buf[32][32 + 1];
    const int nwarps = blockDim.x >> 5;
    const int warp_id = threadIdx.x >> 5;
    const int lane = threadIdx.x & 31;

    // Transpose
    int xmax = (N + 32 - 1) / 32;
    int ymax = (M + 32 - 1) / 32;
    for (int ti = blockIdx.x; ti < xmax * ymax; ti += gridDim.x) {
        int tx = ti % xmax;
        int ty = ti / xmax;
        __syncthreads();

        for (int y = warp_id; y < 32; y += nwarps) {
            int my_x = tx * 32 + lane;
            int my_y = ty * 32 + y;
            if (my_x < N) {
                s_buf[y][lane] = src[N * my_y + my_x];
            }
        }
        __syncthreads();

        for (int x = warp_id; x < 32; x += nwarps) {
            int my_x = ty * 32 + lane;
            int my_y = tx * 32 + x;
            if (my_y < N) {
                dst[M * my_y + my_x] = s_buf[lane][x];
            }
        }
    }
}


#define BATCH 4

__global__ void
nestedLoopJoinCounter_kernel(int* d_signatureMatrix, int rSize, int sSize, int numBins, int *d_resultPositions)
{
    __shared__ int s_signatures[BATCH][NUM_BINS];
    __shared__ int s_counts[BATCH];
    // rSsize needs to be divisible by BATCH
    for (int r = BATCH * blockIdx.x; r < rSize; r += BATCH * gridDim.x) {
        __syncthreads();
        if (threadIdx.x < BATCH) {
            s_counts[threadIdx.x] = 0;
        }
        for (int i = threadIdx.x; i < NUM_BINS; i += blockDim.x) {
#pragma unroll
            for (int j = 0; j < BATCH; ++j) {
	      s_signatures[j][i] = d_signatureMatrix[(r + j) * NUM_BINS + i];
            }
        }
        __syncthreads();

        int my_counts[BATCH];
#pragma unroll
        for (int i = 0; i < BATCH; ++i) {
            my_counts[i] = 0;
        }
        for (int s = threadIdx.x; s < sSize; s += blockDim.x) {
            int identicalMinhashes[BATCH];
            int emptyBins[BATCH];
            for (int i = 0; i < BATCH; ++i) {
                identicalMinhashes[i] = 0;
                emptyBins[i] = 0;
            }
            for (int i = 0; i < NUM_BINS; ++i) {
	      // int ss = d_signatureMatrix[i * (rSize + sSize) + rSize + s];
	      int ss = d_signatureMatrix[rSize * NUM_BINS + i * sSize + s];
#pragma unroll
                for (int j = 0; j < BATCH; ++j) {
                    if (s_signatures[j][i] == ss) {
                        if (s_signatures[j][i] == INT_MAX) {
                            ++emptyBins[j];
                        } else {
                            ++identicalMinhashes[j];
                        }
                    }
                }
            }
#pragma unroll
            for (int i = 0; i < BATCH; ++i) {
                float similarity = (identicalMinhashes[i] * 1.0f) / ((numBins * 1.0f) - (emptyBins[i] * 1.0f));
                if (similarity >= SIMILARITY_THRESHOLD) {
                    ++my_counts[i];
                }
            }
        }
#pragma unroll
        for (int i = 0; i < BATCH; ++i) {
            atomicAdd(&s_counts[i], my_counts[i]);
        }
        __syncthreads();

        if (threadIdx.x < BATCH) {
            atomicAdd(&d_similarPairsCount, s_counts[threadIdx.x]);
            d_resultPositions[r + threadIdx.x] = s_counts[threadIdx.x] * 2;
        }
    }
}

__global__ void
nestedLoopJoinOutputter_kernel(int* d_signatureMatrix, int rSize, int sSize, int numBins, int *d_resultPairs, int *d_resultPositions)
{
    __shared__ int s_signatures[BATCH][NUM_BINS];
    __shared__ int s_offsets[BATCH];
    for (int r = BATCH * blockIdx.x; r < rSize; r += BATCH * gridDim.x) {
        __syncthreads();
        if (threadIdx.x < BATCH) {
            s_offsets[threadIdx.x] = d_resultPositions[r + threadIdx.x];
        }
        for (int i = threadIdx.x; i < NUM_BINS; i += blockDim.x) {
#pragma unroll
            for (int j = 0; j < BATCH; ++j) {
	      // s_signatures[j][i] = d_signatureMatrix[i * (rSize + sSize) + r + j];
	      s_signatures[j][i] = d_signatureMatrix[(r + j) * NUM_BINS + i];
            }
        }
        __syncthreads();

        for (int s = threadIdx.x; s < sSize; s += blockDim.x) {
            int identicalMinhashes[BATCH];
            int emptyBins[BATCH];
#pragma unroll
            for (int i = 0; i < BATCH; ++i) {
                identicalMinhashes[i] = 0;
                emptyBins[i] = 0;
            }
            for (int i = 0; i < NUM_BINS; ++i) {
              //  int ss = d_signatureMatrix[i * (rSize + sSize) + rSize + s]
	      int ss = d_signatureMatrix[rSize * NUM_BINS + i * sSize + s];
#pragma unroll
                for (int j = 0; j < BATCH; ++j) {
                    if (s_signatures[j][i] == ss) {
                        if (s_signatures[j][i] == INT_MAX) {
                            ++emptyBins[j];
                        } else {
                            ++identicalMinhashes[j];
                        }
                    }
                }
            }
#pragma unroll
            for (int i = 0; i < BATCH; ++i) {
                float similarity = (identicalMinhashes[i] * 1.0f) / ((numBins * 1.0f) - (emptyBins[i] * 1.0f));
                if (similarity >= SIMILARITY_THRESHOLD) {
                    int offset = atomicAdd(&s_offsets[i], 2);
                    d_resultPairs[offset] = r + i;
                    d_resultPairs[offset + 1] = s;
                }
            }
        }
    }
}

std::vector<int>
kernelManager(std::vector<int> &h_signatureMatrix, ccsMatrix* h_characteristicMatrix, int numShingles, int sSize, int rSize, int numBins, double &tMinhash, double &tJoin, double &tMemoryTransfer)
{
  int numberOfBlocks, numSets = rSize + sSize, h_similarPairsCount;

  //Device variables
  int *d_signatureMatrix, *d_cmRowIdx, *d_cmColPtr, *d_resultPairs, *d_resultPositions;

  //Size of data structures
  int cmRowIdxSize = h_characteristicMatrix -> row_ind.size();
  int cmColPtrSize = h_characteristicMatrix -> col_ptr.size();
  int smSize = h_signatureMatrix.size();

  //Characteristic matrix
  std::vector<int> h_cmRowIdx = h_characteristicMatrix -> row_ind;
  std::vector<int> h_cmColPtr = h_characteristicMatrix -> col_ptr;

//Timer initialization
  hipEvent_t startMinhash, stopMinhash, startJoin, stopJoin, startInitialTransfer, stopInitialTransfer, startFinalTransfer, stopFinalTransfer;
  float timeMinhash, timeJoin, timeInitialTransfer, timeFinalTransfer;

  //Memory Allocation and Memory transfer CPU -> GPU
  hipEventCreate(&startInitialTransfer);
  hipEventCreate(&stopInitialTransfer);
  hipEventRecord(startInitialTransfer, 0);

  hipMalloc(&d_cmRowIdx, sizeof(int) * cmRowIdxSize);
  hipMalloc(&d_cmColPtr, sizeof(int) * cmColPtrSize);
  hipMalloc(&d_signatureMatrix, sizeof(int) * smSize);
  hipMalloc(&d_resultPositions, sizeof(int) * rSize);

  hipMemcpy(d_cmRowIdx, &h_cmRowIdx[0], sizeof(int) * cmRowIdxSize, hipMemcpyHostToDevice);
  hipMemcpy(d_cmColPtr, &h_cmColPtr[0], sizeof(int) * cmColPtrSize, hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  hipEventRecord(stopInitialTransfer, 0);
  hipEventSynchronize(stopInitialTransfer);
  hipEventElapsedTime(&timeInitialTransfer, startInitialTransfer, stopInitialTransfer);

  //Minhash building signature matrix
  numberOfBlocks = numSets/THREADS_PER_BLOCK;
  if (numSets % THREADS_PER_BLOCK) numberOfBlocks++;
  hipEventCreate(&startMinhash);
  hipEventCreate(&stopMinhash);
  hipEventRecord(startMinhash, 0);

  //computeSignatureMatrix_kernel<<<numberOfBlocks, THREADS_PER_BLOCK>>>(d_signatureMatrix, d_cmRowIdx, d_cmColPtr, numShingles, numSets, numBins);
  computeSignatureMatrix_kernel<<<14 * 7 * 3 * 5, THREADS_PER_BLOCK>>>(d_signatureMatrix, d_cmRowIdx, d_cmColPtr, numShingles, rSize, sSize);

  // hipMemcpy(&h_signatureMatrix[0], d_signatureMatrix, sizeof(int)*smSize, hipMemcpyDeviceToHost);

  // for (int i = 0; i < rSize+sSize; i++) {
  //   for (int j = 0; j < NUM_BINS; j++ ) {
  //     std::cout << h_signatureMatrix[i*NUM_BINS+j] << " ";
  //   }
  //   std::cout << std::endl;
  // }

  hipDeviceSynchronize();
  hipEventRecord(stopMinhash, 0);
  hipEventSynchronize(stopMinhash);
  hipEventElapsedTime(&timeMinhash, startMinhash, stopMinhash);
  tMinhash = timeMinhash*1e-3;

  //Delete the characteristic after the signature matrix constructed
  hipFree(d_cmRowIdx);
  hipFree(d_cmColPtr);

  //Nested Loop Join
  hipEventCreate(&startJoin);
  hipEventCreate(&stopJoin);
  hipEventRecord(startJoin, 0);

  int *d_signatureMatrixT;
  hipMalloc(&d_signatureMatrixT, sizeof(int) * smSize);
  hipMemcpy(d_signatureMatrixT, d_signatureMatrix, sizeof(int) * rSize * NUM_BINS, hipMemcpyDeviceToDevice);
  transpose<<<256, 256>>>(d_signatureMatrix + rSize * NUM_BINS, d_signatureMatrixT + rSize * NUM_BINS, sSize, NUM_BINS);
  nestedLoopJoinCounter_kernel<<<14 * 7 * 3 * 5, THREADS_PER_BLOCK>>>(d_signatureMatrixT, rSize, sSize, numBins, d_resultPositions);
  // nestedLoopJoinCounter_kernel<<<numberOfBlocks, THREADS_PER_BLOCK>>>(d_signatureMatrix, rSize, sSize, numBins, d_resultPositions);
  // std::vector<int> h_resultPositions(rSize);
  // hipMemcpy(&h_resultPositions[0], d_resultPositions, sizeof(int)*rSize, hipMemcpyDeviceToHost);
  // for (int i = 0; i < h_resultPositions.size(); i++) {
  //   std::cout << h_resultPositions[i] << " ";
  // }
  // std::cout << std::endl;
  // std::cout << std::endl;
  // std::cout << std::endl;

  thrust::exclusive_scan(thrust::device, d_resultPositions, d_resultPositions + rSize, d_resultPositions); // in-place scan

  // hipMemcpy(&h_resultPositions[0], d_resultPositions, sizeof(int)*rSize, hipMemcpyDeviceToHost);
  // for (int i = 0; i < h_resultPositions.size(); i++) {
  //   std::cout << h_resultPositions[i] << " ";
  // }
  // std::cout << std::endl;

  hipMemcpyFromSymbol(&h_similarPairsCount, HIP_SYMBOL(d_similarPairsCount), sizeof(int), 0, hipMemcpyDeviceToHost);
  hipMalloc(&d_resultPairs, sizeof(int) * h_similarPairsCount * 2);
  nestedLoopJoinOutputter_kernel<<<14 * 7 * 3 * 5, THREADS_PER_BLOCK>>>(d_signatureMatrixT, rSize, sSize, numBins, d_resultPairs, d_resultPositions);

  hipDeviceSynchronize();
  hipEventRecord(stopJoin, 0);
  hipEventSynchronize(stopJoin);
  hipEventElapsedTime(&timeJoin, startJoin, stopJoin);
  tJoin = timeJoin*1e-3;

  //Transfer memory back to the CPU and free GPU memory
  hipEventCreate(&startFinalTransfer);
  hipEventCreate(&stopFinalTransfer);
  hipEventRecord(startFinalTransfer, 0);

  std::vector<int> h_resultPairs (h_similarPairsCount*2);
  hipMemcpy(&h_resultPairs[0], d_resultPairs, sizeof(int) * h_similarPairsCount * 2, hipMemcpyDeviceToHost);
  //  hipMemcpy(&h_signatureMatrix[0], d_signatureMatrix, sizeof(int)*smSize, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
  hipEventRecord(stopFinalTransfer, 0);
  hipEventSynchronize(stopFinalTransfer);
  hipEventElapsedTime(&timeFinalTransfer, startFinalTransfer, stopFinalTransfer);
  tMemoryTransfer = timeInitialTransfer*1e-3 + timeFinalTransfer*1e-3;

  hipFree(d_signatureMatrix);
  hipFree(d_resultPairs);
  hipFree(d_resultPositions);

  return h_resultPairs;
}
